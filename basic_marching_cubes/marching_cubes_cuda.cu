#include "hip/hip_runtime.h"
#include "marching_cubes.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ unsigned char findCubeIndex(uint3 gridSize, float* f, unsigned int i, unsigned int j, unsigned int k, float isoValue, float* gridValue)
{
	unsigned int cube_vid[8];
	cube_vid[0] = (unsigned int)k*gridSize.y*gridSize.x + j * gridSize.x + i;
	cube_vid[1] = cube_vid[0] + 1;
	cube_vid[2] = cube_vid[0] + 1 + gridSize.x;
	cube_vid[3] = cube_vid[0] + gridSize.x;
	cube_vid[4] = cube_vid[0] + gridSize.x*gridSize.y;
	cube_vid[5] = cube_vid[1] + gridSize.x*gridSize.y;
	cube_vid[6] = cube_vid[2] + gridSize.x*gridSize.y;
	cube_vid[7] = cube_vid[3] + gridSize.x*gridSize.y;

	gridValue[0] = f[cube_vid[0]];
	gridValue[1] = f[cube_vid[1]];
	gridValue[2] = f[cube_vid[2]];
	gridValue[3] = f[cube_vid[3]];
	gridValue[4] = f[cube_vid[4]];
	gridValue[5] = f[cube_vid[5]];
	gridValue[6] = f[cube_vid[6]];
	gridValue[7] = f[cube_vid[7]];

	unsigned char cubeindex = 0;
	if (f[cube_vid[0]] < isoValue) cubeindex |= 1;
	if (f[cube_vid[1]] < isoValue) cubeindex |= 2;
	if (f[cube_vid[2]] < isoValue) cubeindex |= 4;
	if (f[cube_vid[3]] < isoValue) cubeindex |= 8;
	if (f[cube_vid[4]] < isoValue) cubeindex |= 16;
	if (f[cube_vid[5]] < isoValue) cubeindex |= 32;
	if (f[cube_vid[6]] < isoValue) cubeindex |= 64;
	if (f[cube_vid[7]] < isoValue) cubeindex |= 128;

	return cubeindex;
}

__device__ float3 vertexInterp(float isoValue, float3 p1, float3 p2, float valp1, float valp2, float eps)
{
	float mu;
	float3 p;

	if (fabs(isoValue - valp1) < eps) //0.00001)
		return p1;
	if (fabs(isoValue - valp2) < eps) //0.00001)
		return p2;
	if (fabs(valp1 - valp2) < eps) //0.00001)
		return p1;

	mu = (isoValue - valp1) / (valp2 - valp1);

	p.x = p1.x + mu * (p2.x - p1.x);
	p.y = p1.y + mu * (p2.y - p1.y);
	p.z = p1.z + mu * (p2.z - p1.z);

	return p;
}

__global__ void kernel_extract_mesh(unsigned int nx, unsigned int ny, unsigned int nz, unsigned int numVoxels,
	float bmin_x, float bmin_y, float bmin_z, float dx, float dy, float dz,
	float* f, int* edgeTable, int* triTable,
	unsigned int* vertIndex, unsigned int* triIndex, float* vertex, unsigned int* triangle,
	float isoValue, float eps, float CUT_VAL)
{
	unsigned int tid = threadIdx.x + (unsigned int)blockIdx.x*blockDim.x;
	if (tid < numVoxels)
	{
		float3 vertlist[12];
		float gv[8];
		float3 gp[8];

		unsigned int i = tid % nx;
		unsigned int j = ((tid - i) / nx) % ny;
		unsigned int k = ((tid - i) / nx) / ny;

		uint3 gridSize = make_uint3(nx + 1, ny + 1, nz + 1);

		// Find the cube index for the current voxel
		unsigned char cubeindex = findCubeIndex(gridSize, f, i, j, k, isoValue, gv);

		if (fabs(gv[0]) < CUT_VAL && fabs(gv[1]) < CUT_VAL && fabs(gv[2]) && CUT_VAL &&
			fabs(gv[3]) < CUT_VAL && fabs(gv[4]) < CUT_VAL && fabs(gv[5]) && CUT_VAL &&
			fabs(gv[6]) < CUT_VAL && fabs(gv[7]) < CUT_VAL &&
			edgeTable[cubeindex] > 0)
		{
			// Now, the current is a surface cell
			//std::fill(vertlist.begin(), vertlist.end(), mg::Vector3f(0.0, 0.0, 0.0));
			float3 bmin = make_float3(bmin_x, bmin_y, bmin_z);

			gp[0].x = bmin.x + i * dx;
			gp[0].y = bmin.y + j * dy;
			gp[0].z = bmin.z + k * dz;	// make_float3(i*dx, j*dy, k*dz);

			gp[1].x = bmin.x + (i + 1) * dx;
			gp[1].y = bmin.y + j * dy;
			gp[1].z = bmin.z + k * dz;  //gp[1] = bmin + make_float3((i + 1)*dx, j*dy, k*dz);

			gp[2].x = bmin.x + (i + 1) * dx;
			gp[2].y = bmin.y + (j + 1) * dy;
			gp[2].z = bmin.z + k * dz;	//gp[2] = bmin + make_float3((i + 1)*dx, (j + 1)*dy, k*dz);

			gp[3].x = bmin.x + i * dx;
			gp[3].y = bmin.y + (j + 1) * dy;
			gp[3].z = bmin.z + k * dz;	//gp[3] = bmin + make_float3(i*dx, (j + 1)*dy, k*dz);

			gp[4].x = gp[0].x;
			gp[4].y = gp[0].y;
			gp[4].z = gp[0].z + dz;        //gp[4] = gp[0] + make_float3(0.0f, 0.0f, dz);

			gp[5].x = gp[1].x;
			gp[5].y = gp[1].y;
			gp[5].z = gp[1].z + dz;			//gp[5] = gp[1] + make_float3(0.0f, 0.0f, dz);

			gp[6].x = gp[2].x;
			gp[6].y = gp[2].y;
			gp[6].z = gp[2].z + dz;			//gp[6] = gp[2] + make_float3(0.0f, 0.0f, dz);

			gp[7].x = gp[3].x;
			gp[7].y = gp[3].y;
			gp[7].z = gp[3].z + dz;		 //gp[7] = gp[3] + make_float3(0.0f, 0.0f, dz);

			// Create vertices on edges
			if (edgeTable[cubeindex] & 1)		vertlist[0] = vertexInterp(isoValue, gp[0], gp[1], gv[0], gv[1], eps);
			if (edgeTable[cubeindex] & 2)		vertlist[1] = vertexInterp(isoValue, gp[1], gp[2], gv[1], gv[2], eps);
			if (edgeTable[cubeindex] & 4)		vertlist[2] = vertexInterp(isoValue, gp[2], gp[3], gv[2], gv[3], eps);
			if (edgeTable[cubeindex] & 8)		vertlist[3] = vertexInterp(isoValue, gp[3], gp[0], gv[3], gv[0], eps);
			if (edgeTable[cubeindex] & 16)		vertlist[4] = vertexInterp(isoValue, gp[4], gp[5], gv[4], gv[5], eps);
			if (edgeTable[cubeindex] & 32)		vertlist[5] = vertexInterp(isoValue, gp[5], gp[6], gv[5], gv[6], eps);
			if (edgeTable[cubeindex] & 64)		vertlist[6] = vertexInterp(isoValue, gp[6], gp[7], gv[6], gv[7], eps);
			if (edgeTable[cubeindex] & 128)		vertlist[7] = vertexInterp(isoValue, gp[7], gp[4], gv[7], gv[4], eps);
			if (edgeTable[cubeindex] & 256)		vertlist[8] = vertexInterp(isoValue, gp[0], gp[4], gv[0], gv[4], eps);
			if (edgeTable[cubeindex] & 512)		vertlist[9] = vertexInterp(isoValue, gp[1], gp[5], gv[1], gv[5], eps);
			if (edgeTable[cubeindex] & 1024)	vertlist[10] = vertexInterp(isoValue, gp[2], gp[6], gv[2], gv[6], eps);
			if (edgeTable[cubeindex] & 2048)	vertlist[11] = vertexInterp(isoValue, gp[3], gp[7], gv[3], gv[7], eps);

			// Generate triangles
			for (int ti = 0; triTable[cubeindex * 16 + ti] != -1; ti += 3) {
				//vertex_.resize(vertexIndex + 3);
				int vid = atomicAdd(vertIndex, 3);
				vertex[3 * (vid + 0) + 0] = vertlist[triTable[cubeindex * 16 + ti]].x;
				vertex[3 * (vid + 0) + 1] = vertlist[triTable[cubeindex * 16 + ti]].y;
				vertex[3 * (vid + 0) + 2] = vertlist[triTable[cubeindex * 16 + ti]].z;

				vertex[3 * (vid + 1) + 0] = vertlist[triTable[cubeindex * 16 + ti + 1]].x;
				vertex[3 * (vid + 1) + 1] = vertlist[triTable[cubeindex * 16 + ti + 1]].y;
				vertex[3 * (vid + 1) + 2] = vertlist[triTable[cubeindex * 16 + ti + 1]].z;

				vertex[3 * (vid + 2) + 0] = vertlist[triTable[cubeindex * 16 + ti + 2]].x;
				vertex[3 * (vid + 2) + 1] = vertlist[triTable[cubeindex * 16 + ti + 2]].y;
				vertex[3 * (vid + 2) + 2] = vertlist[triTable[cubeindex * 16 + ti + 2]].z;

				//triangle_.resize(triIndex + 1);
				int tid = atomicAdd(triIndex, 1);
				triangle[3 * tid + 0] = vid;
				triangle[3 * tid + 1] = vid + 2;
				triangle[3 * tid + 2] = vid + 1;

				//if (vid == 0)
				//{
				//	printf("vertex[0] : %f %f %f\n", vertex[vid+0], vertex[vid+1], vertex[vid+2]);
				//	printf("triangle[0] : %d %d %d\n", triangle[0], triangle[1], triangle[2]);
				//}
			}
		}



		//if (f[tid] > -1.0f && f[tid] < 1.0f)
		//{
		//	atomicAdd(sum, f[tid]);
			//if (sum[0] < 10.0f)
			//if(triIndex[0] == 0)
			//{
			//	printf("sum=%f, hurei\n", sum[0]);
				//printf("vertlist[0] : %f %f %f\n", vertlist[0].x, vertlist[0].y, vertlist[0].z);
			//}
		//}


	}
}

void MarchingCubes::generateSurfaceMesh_cuda(float isoValue, float cutValue, const char* filename)
{
	const unsigned int nx = volumeSize_.x;
	const unsigned int ny = volumeSize_.y;
	const unsigned int nz = volumeSize_.z;

	const unsigned int gnx = nx + 1;
	const unsigned int gny = ny + 1;
	const unsigned int gnz = nz + 1;

	const float dx = voxelSize_.x;
	const float dy = voxelSize_.y;
	const float dz = voxelSize_.z;

	unsigned int numVoxels = (unsigned int)nx * ny * nz;

	// Device data allocation
	float* f_d;
	int* edgeTable_d;
	int* triTable_d;
	unsigned int* vertIndex_d;
	unsigned int* triIndex_d;
	float* vertex_d;
	unsigned int* triangle_d;
	float * vertex;
	unsigned int* triangle;

	if (hipMalloc(&f_d, sizeof(float)*gnx*gny*gnz) != hipSuccess) { printf("allocation error : f_d\n"); }
	if(hipMalloc(&edgeTable_d, sizeof(int) * 256) != hipSuccess) { printf("allocation error : edgeTable_d\n"); }
	if(hipMalloc(&triTable_d, sizeof(int) * 256 * 16) != hipSuccess) { printf("allocation error : triTable_d\n"); }
	if(hipMalloc(&vertIndex_d, sizeof(unsigned int)) != hipSuccess) { printf("allocation error : vertIndex_d\n"); }
	if(hipMalloc(&triIndex_d, sizeof(unsigned int)) != hipSuccess) { printf("allocation error : triIndex_d\n"); }

	// TODO : The following two numbers need to be set properly.
	int maxNumVertex = 1028 * 1028;
	int maxNumTriangles = maxNumVertex;
	vertex_.resize(maxNumVertex);
	triangle_.resize(maxNumTriangles);
	if(hipMalloc(&vertex_d, sizeof(float) * 3 * maxNumVertex) != hipSuccess) { printf("allocation error : vertex_d\n"); }
	if(hipMalloc(&triangle_d, sizeof(unsigned int) * 3 * maxNumTriangles) != hipSuccess) { printf("allocation error : triangle_d\n"); }
	hipHostMalloc(&vertex, sizeof(float) * 3 * maxNumVertex);
	hipHostMalloc(&triangle, sizeof(unsigned int) * 3 * maxNumTriangles);

	// Copy data from host to device
	if(hipMemcpy(f_d, f_.data(), sizeof(float)*gnx*gny*gnz, hipMemcpyHostToDevice) != hipSuccess) { printf("copy error : f_d\n"); }
	if(hipMemcpy(edgeTable_d, edgeTable_, sizeof(int) * 256, hipMemcpyHostToDevice) != hipSuccess) { printf("copy error : edgeTable_d\n"); }
	if(hipMemcpy(triTable_d, triTable_, sizeof(int) * 256 * 16, hipMemcpyHostToDevice) != hipSuccess) { printf("copy error : triTable_d\n"); }

	unsigned int triIndex = 0;
	unsigned int vertIndex = 0;
	if(hipMemcpy(vertIndex_d, &vertIndex, sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) { printf("copy error : vertIndex_d\n"); }
	if(hipMemcpy(triIndex_d, &triIndex, sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) { printf("allocation error : triIndex_d\n"); }

	// Kernel
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	const int nthreads = 512;
	int nblocks = (numVoxels + nthreads - 1) / nthreads;
	kernel_extract_mesh << < nblocks, nthreads >> > (nx, ny, nz, numVoxels, bmin_.x, bmin_.y, bmin_.z, dx, dy, dz, 
		f_d, edgeTable_d, triTable_d,
		vertIndex_d, triIndex_d, vertex_d, triangle_d, isoValue, eps, cutValue);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime = 0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "- Kernel computing : " << elapsedTime << " ms" << std::endl;

	// Copy data from device to host
	hipMemcpy(&vertIndex, vertIndex_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(&triIndex, triIndex_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	std::cout << "- vertexIndex=" << vertIndex << ", triIndex=" << triIndex << std::endl;

	//vertex_.resize(vertIndex);
	//triangle_.resize(triIndex);
	//hipMemcpy(vertex_.data(), vertex_d, sizeof(float) * 3 * (vertIndex), hipMemcpyDeviceToHost);
	//hipMemcpy(triangle_.data(), triangle_d, sizeof(unsigned int) * 3 * (triIndex), hipMemcpyDeviceToHost);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMemcpy(vertex, vertex_d, sizeof(float) * 3 * (vertIndex), hipMemcpyDeviceToHost);
	hipMemcpy(triangle, triangle_d, sizeof(unsigned int) * 3 * (triIndex), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "- Data copying device to host : " << elapsedTime << " ms" << std::endl;

	//vertex_.resize(vertIndex);
	//triangle_.resize(triIndex);
	//memcpy(vertex_.data(), vertex, sizeof(float) * 3 * vertIndex);
	//memcpy(triangle_.data(), triangle, sizeof(unsigned int) * 3 * triIndex);

	exportMeshInObj(filename, vertIndex, vertex, triIndex, triangle);

	// Free memory
	hipFree(f_d);
	hipFree(edgeTable_d);
	hipFree(triTable_d);
	hipFree(vertIndex_d);
	hipFree(triIndex_d);
	hipFree(vertex_d);
	hipFree(triangle_d);
	hipHostFree(vertex);
	hipHostFree(triangle);
}